#include "hip/hip_runtime.h"


namespace summation {

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduce(int *in, int *out, unsigned int n) {
    volatile extern __shared__ int s[];
    unsigned int  tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2)+tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    s[tid] = 0;
    while(i < n) {
        if(nIsPow2 || i + blockSize < n)
            s[tid] += in[i] + in[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    if(blockSize >= 2048) { if(tid < 1024) s[tid] += s[tid+1024]; __syncthreads(); }
    if(blockSize >= 1024) { if(tid <  512) s[tid] += s[tid+ 512]; __syncthreads(); }
    if(blockSize >=  512) { if(tid <  256) s[tid] += s[tid+ 256]; __syncthreads(); }
    if(blockSize >=  256) { if(tid <  128) s[tid] += s[tid+ 128]; __syncthreads(); }
    if(blockSize >=  128) { if(tid <   64) s[tid] += s[tid+  64]; __syncthreads(); }

    if(tid < 32) {
        if (blockSize >= 64) s[tid] += s[tid + 32];
        if (blockSize >= 32) s[tid] += s[tid + 16];
        if (blockSize >= 16) s[tid] += s[tid +  8];
        if (blockSize >=  8) s[tid] += s[tid +  4];
        if (blockSize >=  4) s[tid] += s[tid +  2];
        if (blockSize >=  2) s[tid] += s[tid +  1];
    }
    if(tid == 0) out[blockIdx.x] = s[0];
}

bool isPow2(int size) { return __builtin_popcount(size) == 1; }

void invoke_reduce_kernel(int *in, int *out, int blocks, int threads, int n) {
    int shared = (threads <= 32) ? 2 * sizeof(int) * threads : sizeof(int) * threads;
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);


    if (isPow2(n)) {
        switch(threads) {
        case 2048: reduce<2048, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case 1024: reduce<1024, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  512: reduce< 512, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  256: reduce< 256, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  128: reduce< 128, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   64: reduce<  64, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   32: reduce<  32, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   16: reduce<  16, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    8: reduce<   8, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    4: reduce<   4, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    2: reduce<   2, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    1: reduce<   1, true><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        }
    } else {
        switch(threads) {
        case 2048: reduce<2048, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case 1024: reduce<1024, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  512: reduce< 512, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  256: reduce< 256, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case  128: reduce< 128, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   64: reduce<  64, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   32: reduce<  32, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case   16: reduce<  16, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    8: reduce<   8, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    4: reduce<   4, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    2: reduce<   2, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        case    1: reduce<   1, false><<<dimGrid, dimBlock, shared>>>(in, out, n); break;
        }
    }
}

#define NPB (1024)  // number of calculation per block
#define NT (NPB/2) // number of thread

int calc(int *A, int n) {
    int blocks = n/NPB, threads = NT, result;
    while(blocks) {
        invoke_reduce_kernel(A, A, blocks, threads, n);
        blocks /= NPB;
    }
    hipMemcpy(&result, &A[0], sizeof(int), hipMemcpyDeviceToHost);
    return result;
}
}
