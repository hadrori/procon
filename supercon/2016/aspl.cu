#linclude "summation.cu"

namespace aspl {

int calc(int *A, int n) {
    int *B;
    hipMalloc(&B, sizeof(int)*n*n);
    hipMemcpy(B, A, sizeof(int)*n*n, hipMemcpyHostToDevice);
    apsp(B, n, n);
    return summation::calc(B, n*n);
}
}
