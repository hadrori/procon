#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "macro.c"

#define NBS_X 64
#define NBS_Y 8

__global__ void naive_kernel(g_type *A, int n, int all_w) {
    const int i0 = blockIdx.y * blockDim.y + threadIdx.y;
    const int j  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ws = NAIVE_BLOCK_SIZE_X + 1;
    __shared__ g_type B[NAIVE_BLOCK_SIZE_X * ws];
    for(int i = i0; i < n; i += blockDim.y)
        B[i * ws + j] = A[i * all_w + j];
    __syncthreads();
    rep(k,n) {
        for(int i = i0; i < n; i += blockDim.y)
            B[i*ws+j] = min(B[i*ws+j], B[i*ws+k] + B[k*ws+j]);
        __syncthreads();
    }
    for(int i = i0; i < n; i += blockDim.y) A[i*all_w+j] = B[i*ws+j];
}

void naive(int *A, int n, int all_w) {
    const dim3 grid(n/NBS_X, n/NBS_Y), block(NBS_X, NBS_Y);
    naive_kernel<<<grid, block>>>(A, n, all_w);
}

#define BS 256
#define B_W 64
#define B_H 64

__global__ void mul_kernel(g_type *C, g_type *A, g_type *B, int n, int all_w) {
    const int i = blockIdx.y * blockDim.y + threadIdx.y,
              j = blockIdx.x * blockDim.x + threadIdx.x;
    rep(k,n) C[i*n+j] = min(C[i*n+j], A[i*n+k]+B[k*n+j]);
}

void mul(g_type *C, g_type *A, g_type *B, int n, int all_w) {
    const gim3 grid(n/B_W, n/B_H), block(BS);
    mul_kernel<<<grid, block>>>(C, A, B, n, all_w);
}

void apsp(g_type *A, int n, int all_w) {
    if (n <= NBS_X) {
        naive(A, n, all_w);
    }
    else {
        const int h = n/2, a11 = 0, a12 = h, a21 = h*all_w, a22 = a21 + h;
        apsp(A+a11, h, all_w);
        mul(A+a12, A+a11, A+a12, h, all_w);
        mul(A+a21, A+a21, A+a11, h, all_w);
        mul(A+a22, A+a21, A+a12, h, all_w);
        apsp(A+a22, h, all_w);
        mul(A+a21, A+a22, A+a21, h, all_w);
        mul(A+a12, A+a12, A+a22, h, all_w);
        mul(A+a11, A+a12, A+a21, h, all_w);
    }
}

g_type sum(g_type *A) {
}

g_type aspl(g_type *A, int n) {
    g_type *B;
    hipMalloc(&B, sizeof(g_type)*n*n);
    hipMemcpy(B, A, sizeof(g_type)*n*n, hipMemcpyHostToDevice);
    apsp(B, n, n);
    g_type ret = sum(B, n);
    hipFree(B);
    return ret;
}
